#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<time.h>
#include <stdio.h>

void printMatrix(double* mat, int m, int n) {
    printf("\t\t");
    for (int i = 0; i < m * n; i++)
    {
        if (mat[i] < 0) {
            printf("%4.1f\t", mat[i]);
        }
        else {
            printf("%4.2f\t", mat[i]);
        }
        if ((i + 1) % n == 0) {
            printf("\n\t\t");
        }
    }
    printf("\n");
}

void MatrixMultiplication(double* A, double* B, double* C, int N)
{
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double sum = 0;
            for (int k = 0; k < N; k++)
                sum += A[i * N + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
    }
}

__global__ void multKernel1(double* c, double* a, double* b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

__global__ void multKernel2(double* c, double* a, double* b)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    int k = threadIdx.y;

    int l = blockIdx.x;
    int m = blockIdx.y;
    int n = blockIdx.y;

    printf("Indice del hilo: (%d, %d, %d) | Indice del bloque: (%d, %d, %d) | Calculando el producto: {%f} * {%f}\n", i, j, k, l, m, n, a[i], b[i]);
    //printf("Calculando el producto: {%f} * {%f}\n", a[i], b[i]);
    c[i] = a[i] * b[i];
}

__global__ void MatrixMultiplicationCuda(double* c, double* a, double* b, int N)
{
    int i = threadIdx.x; 
    int j = threadIdx.y;

    double sum = 0;
    for (int k = 0; k < N; k++) {
        sum += a[i * N + k] * b[k * N + j];
    }
    c[i * N + j] = sum;
      

}

hipError_t multWithCuda1(double* c, double* a, double* b, unsigned int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multKernel1 << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

hipError_t multWithCuda2(double* c, double* a, double* b, unsigned int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    multKernel2 << <10, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multKernel2 launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

hipError_t multMatrixwithCuda(double* c, double* a, double* b, unsigned int size)
{
    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    int N = size * size;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, N * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(size, size);
    MatrixMultiplicationCuda <<<1, threadsPerBlock>>> (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "MatrixMultiplicationCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, N * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int funcion1() 
{
    const int arraySize = 100;
    double a[arraySize];
    double b[arraySize];
    double c[arraySize];

    for (int i = 0; i < arraySize; i++) {
        a[i] = (double)i;
        b[i] = (double)i * 2;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = multWithCuda1(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multWithCuda failed!");
        return 1;
    }

    double prod = 0;
    for (int i = 0; i < arraySize; i++) {
        prod += c[i];
    }

    printf("Producto escalar: %f", prod);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

int funcion2() 
{
    const int arraySize = 100;
    double a[arraySize];
    double b[arraySize];
    double c[arraySize];

    for (int i = 0; i < arraySize; i++) {
        a[i] = (double)i;
        b[i] = (double)i * 2;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = multWithCuda2(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multWithCuda failed!");
        return 1;
    }
    
    double prod = 0;
    for (int i = 0; i < arraySize; i++) {
        prod += c[i];
    }

    printf("Producto escalar: %f", prod);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

int funcion3()
{
    const int N = 3;

    double a[N * N];
    double b[N * N];
    double c[N * N];

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
            b[i * N + j] = i - j;
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double sum = 0;
            for (int k = 0; k < N; k++)
                sum += a[i * N + k] * b[k * N + j];
            c[i * N + j] = sum;
        }
    }

    printf("Matriz A (suma de su numero de fila mas su numero de columna):\n");
    printMatrix(a, 3, 3);
    printf("Matriz B (resta de su numero de fila menos su numero de columna):\n");
    printMatrix(b, 3, 3);
    printf("Resultado de la multiplicacion de A * B:\n");
    printMatrix(c, 3, 3);

    return 0;
}

int funcion4()
{
    const int N = 4;

    double a[N * N];
    double b[N * N];
    double c[N * N];

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
            b[i * N + j] = i - j;
        }
    }

    clock_t start, stop;
    start = clock(); 
    for (int i = 0; i < 1000000; i++) {
        MatrixMultiplication(a, b, c, N);
    }
    stop = clock();

    printf("Matriz A (suma de su numero de fila mas su numero de columna):\n");
    printMatrix(a, N, N);
    printf("Matriz B (resta de su numero de fila menos su numero de columna):\n");
    printMatrix(b, N, N);
    printf("Resultado de la multiplicacion de A * B:\n");
    printMatrix(c, N, N);
    printf("Tiempo secuencial: %4.8f segundos\n", (double)(stop - start) / CLOCKS_PER_SEC / 1000000);

    return 0;
}

int funcion5()
{
    const int N = 3;

    double a[N * N];
    double b[N * N];
    double c[N * N];

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
            b[i * N + j] = i - j;
        }
    }

    hipError_t cudaStatus = multMatrixwithCuda(c, a, b, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multWithCuda failed!");
        return 1;
    }

    printf("Matriz A (suma de su numero de fila mas su numero de columna):\n");
    printMatrix(a, N, N);
    printf("Matriz B (resta de su numero de fila menos su numero de columna):\n");
    printMatrix(b, N, N);
    printf("Resultado de la multiplicacion de A * B:\n");
    printMatrix(c, N, N);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

int main()
{
    int error = 0;
    //error = funcion1();
    //error = funcion2();
    //error = funcion3();
    //error = funcion4();
    error = funcion5();

    return error;
}